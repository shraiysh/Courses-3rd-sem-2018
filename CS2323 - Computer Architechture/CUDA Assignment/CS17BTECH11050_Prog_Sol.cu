#define N 512
#define NUM_BLOCKS 16
#define NUM_THREADS 48
//Do not change above three lines.

//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated. 

/*Remember the following guidelines to avoid losing marks
This exercise is quite simple. The only tricky part is that total number of threads (NUM_BLOCKS*NUM_THREADS) may be different (higher or lower) from N.

Index of an array should not exceed the array size. 

No output array-element should be computed more than once

No marks will be given if the program does not compile or run (TAs will not debug your program at all)

Do not change the name of any variable that we have introduced. 
*/



#include <hip/hip_runtime.h>
#include <stdio.h>

//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than array-element or not process any array-element at all. 
__global__ void addElements( int* a, int* b, int* c) {
  int index = NUM_THREADS * blockIdx.x + threadIdx.x;
  for(int i=0;i<N;i++){
    if(index + N*i < N*N){          
      c[index+N*i] = a[index+N*i] + b[index + N*i];
    }
  }
  if(N>NUM_BLOCKS * NUM_THREADS) {
    index += NUM_BLOCKS*NUM_THREADS;
    for(int i=0;i<N;i++) 
      if(index < N) c[index+N*i] = a[index+N*i] + b[index + N*i];
  }
}

int main (int argc, char **argv) {
  int A[N][N], B[N][N], C[N][N];
  int *d_A, *d_B, *d_C; // These are the copies of A, B and C on the GPU
  int *h_C;       // This is a host copy of the output of B from the GPU
  int i, j;

  int size = N * N * sizeof(int);

  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      A[i][j] = i+j;
      B[i][j]= 2*j-1;      
    }
  }

  // sequential implementation of main computation
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      C[i][j] = A[i][j]+B[i][j];
    }
  }

  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A, d_B and d_C
  hipMalloc ( (void **) &d_A, size);
  hipMalloc ( (void **) &d_B, size);
  hipMalloc ( (void **) &d_C, size);
  h_C = (int *)malloc(size);

  // TODO: COPY A TO d_A
  hipMemcpy( d_A, A, size, hipMemcpyHostToDevice );

  // TODO: COPY B TO d_B
  hipMemcpy( d_B, B, size, hipMemcpyHostToDevice );

  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
  addElements <<< NUM_BLOCKS, NUM_THREADS >>> (d_A, d_B, d_C);
   //Use NUM_BLOCKS blocks, each with NUM_THREADS  threads

  // TODO: COPY d_C BACK FROM GPU to CPU in variable h_C
  hipMemcpy( h_C, d_C, size, hipMemcpyDeviceToHost );

  // TODO: Verify result is correct by comparing
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      int index = i*N + j;
      if ( h_C[index] - C[i][j] != 0 )
        printf("i = %d, j = %d\n", i, j);
    //TODO: compare each element of h_C and C by subtracting them
        //print only those elements for which the above subtraction is non-zero
    }
   }
    //IF even one element of h_C and C differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
}
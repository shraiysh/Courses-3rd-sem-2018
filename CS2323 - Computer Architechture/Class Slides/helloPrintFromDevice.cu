
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%d %c\n", threadIdx.x, STR[threadIdx.x % STR_LENGTH]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;
	hello<<<num_blocks,num_threads>>>();
	hipDeviceSynchronize();

	return 0;
}
